
#include <hip/hip_runtime.h>
__device__ unsigned int reduce_sum(unsigned int in)
{
    extern __shared__ unsigned int sdata[];

    // Perform first level of reduction:
    // - Write to shared memory
    unsigned int ltid = threadIdx.x;

    sdata[ltid] = in;
    __syncthreads();

    // Do reduction in shared mem
    for (unsigned int s = blockDim.x / 2 ; s > 0 ; s >>= 1)
    {
        if (ltid < s)
        {
            sdata[ltid] += sdata[ltid + s];
        }

        __syncthreads();
    }

    return sdata[0];
}

__device__ float reduce_sum(float in)
{
    extern __shared__ float sdataF[];

    // Perform first level of reduction:
    // - Write to shared memory
    unsigned int ltid = threadIdx.x;

    sdataF[ltid] = in;
    __syncthreads();

    // Do reduction in shared mem
    for (unsigned int s = blockDim.x / 2 ; s > 0 ; s >>= 1)
    {
        if (ltid < s)
        {
            sdataF[ltid] += sdataF[ltid + s];
        }

        __syncthreads();
    }

    return sdataF[0];
}
