
#include <hip/hip_runtime.h>
__device__ unsigned int reduce_sum(unsigned int in)
{
    extern __shared__ unsigned int sdata[];

    // Perform first level of reduction:
    // - Write to shared memory
    unsigned int ltid = threadIdx.x;

    sdata[ltid] = in;
    __syncthreads();

    // Do reduction in shared mem
    for (unsigned int s = blockDim.x / 2 ; s > 0 ; s >>= 1)
    {
        if (ltid < s)
        {
            sdata[ltid] += sdata[ltid + s];
        }

        __syncthreads();
    }

    return sdata[0];
}
