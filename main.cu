#include <ctime>
#include <functional>
#include <vector>
#include <numeric>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "naive_kernel.hu"
#include "batchrng_kernel.hu"
#include "misc.hu"

double compute_naive(dim3 grid, dim3 block, unsigned int device,
		     unsigned int iterationsperThread);

double compute_batchrng(dim3 grid, dim3 block, unsigned int device,
			unsigned int iterationsperThread,
			hipDeviceProp_t *const deviceProp);

int main (int argc, char ** argv)
{
    unsigned int kernel = 0;
    double piest;
    hipDeviceProp_t deviceProp;
    unsigned int device = 0;
    handleCudaErrors(hipGetDeviceProperties(&deviceProp, device));

    unsigned int iterationsPerThread = 1000 * 1000;
    dim3 grid = 16;
    dim3 block = 64;

    parseArgs(argc, argv, &iterationsPerThread, &deviceProp,
	      &grid.x, &block.x, &kernel);

    switch (kernel) {
    case 0:
	piest = compute_naive(grid, block, device, iterationsPerThread);
	break;
    case 1:
	piest = compute_batchrng(grid, block, device, iterationsPerThread,
				 &deviceProp);
	break;
    }
    
    reportResults(piest, iterationsPerThread, grid.x, block.x, &deviceProp);
    
    return 0;
}

double compute_naive(dim3 grid, dim3 block, unsigned int device,
		     unsigned int iterationsperThread)
{
    handleCudaErrors(hipSetDevice(device));

    hiprandState *d_rngStates = 0;
    handleCudaErrors(hipMalloc((void **) &d_rngStates, grid.x * block.x * sizeof(hiprandState)));

    float *d_res = 0;
    handleCudaErrors(hipMalloc((void **) &d_res, grid.x * sizeof(float)));

    initRNG<<<grid, block>>>(d_rngStates, time(NULL));

    naive_kernel<<<grid, block,  block.x * sizeof(unsigned int)>>>(d_res, d_rngStates, iterationsperThread);

    std::vector<float> res(grid.x);
    handleCudaErrors(hipMemcpy(&res[0], d_res, grid.x * sizeof(float),
				hipMemcpyDeviceToHost));

    double estimate = std::accumulate(res.begin(), res.end(), 0.0);
    hipFree(d_rngStates);
    hipFree(d_res);

    return estimate;
}

double compute_batchrng(dim3 grid, dim3 block, unsigned int device,
			unsigned int its,
			hipDeviceProp_t *const deviceProp)
{
    handleCudaErrors(hipSetDevice(device));
    //Set up the RNG
    using namespace std::placeholders;
    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator, time(NULL));

    auto unifGen = std::bind(hiprandGenerateUniform, generator, _1, _2);
    
    //For partial results
    float *d_res = 0;
    handleCudaErrors(hipMalloc((void **) &d_res, grid.x * sizeof(float)));

    //To calculate the final result
    double runningEstimate = 0;
    
    //Random number vector allocation strategy
    unsigned int numThreads = grid.x * block.x;
    unsigned long int totalSize = sizeof(float) * its * numThreads;
    unsigned long int vecSize = numThreads * 128 * 1024;
    unsigned long int remainSize = totalSize;

    float * d_angleVec = 0;
    handleCudaErrors(hipMalloc((void**) d_angleVec, vecSize));

    float * d_distVec = 0;
    handleCudaErrors(hipMalloc((void**) d_distVec, vecSize));


    unsigned int vecCount = vecSize / sizeof(float);

    //Here we go!
    while (remainSize > sizeof(float)) {
	if (remainSize < vecSize) {
	    vecCount = remainSize / sizeof(float);
	}

	unifGen(d_angleVec, vecCount);
	unifGen(d_distVec, vecCount);

	batchrng_kernel<<<grid, block,  block.x * sizeof(unsigned int)>>>
	    ( d_res, d_angleVec, d_distVec, vecCount);

	std::vector<float> res(grid.x);
	handleCudaErrors(hipMemcpy(&res[0], d_res, grid.x * sizeof(float),
				    hipMemcpyDeviceToHost));

	runningEstimate += std::accumulate(res.begin(), res.end(), 0.0);
	
	if (remainSize > vecSize) {
	    remainSize -= vecSize;
	}
	else {
	    break;
	}
    }

    hipFree(d_angleVec);
    hipFree(d_distVec);
    hipFree(d_res);
    
    return runningEstimate;
}
