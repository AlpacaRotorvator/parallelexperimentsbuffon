#include "hip/hip_runtime.h"
#include <ctime>
#include <hip/hip_runtime.h>
#include "naive_kernel.hu"
#include "misc.hu"

constexpr double pi() { return acos(-1); };

double compute_naive(dim3 grid, dim3 block, unsigned int device,
		     unsigned int iterationsperThread);

int main (int argc, char ** argv)
{
    double piest;
    hipDeviceProp_t deviceProp;
    unsigned int device = 0;
    handleCudaErrors(hipGetDeviceProperties(&deviceProp, device));

    unsigned int iterationsPerThread = 1000 * 1000;
    dim3 grid = 16;
    dim3 block = 64;

    parseArgs(argc, argv, &iterationsPerThread, &deviceProp, &grid.x, &block.x);

    piest = compute_naive(grid, block, device, iterationsPerThread);

    cout << piest;
    return 0;
}

double compute_naive(dim3 grid, dim3 block, unsigned int device,
		     unsigned int iterationsperThread)
{
    handleCudaErrors(hipSetDevice(device));

    hiprandState *d_rngStates = 0;
    handleCudaErrors(hipMalloc((void **) &d_rngStates, grid.x * block.x * sizeof(hiprandState)));

    double *d_res = 0;
    handleCudaErrors(hipMalloc((void **) &d_res, grid.x * sizeof(double)));

    initRNG<<<grid, block>>>(d_rngStates, time(NULL));

    naive_kernel<<<grid, block>>>(d_res, d_rngStates, iterationsperThread);

    std::vector<double> res(grid.x);
    handleCudaErrors(hipMemcpy(&res[0], d_res, grid.x * sizeof(double),
				hipMemcpyDeviceToHost));

    double estimate = std::accumulate(res.begin(), res.end(), 0);
    estimate /= grid.x * block.x;

    hipFree(d_rngStates);
    hipFree(d_res);

    return estimate;
}
