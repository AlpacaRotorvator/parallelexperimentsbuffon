#include <hip/hip_runtime.h>
#include "misc.hxx"


int main (int argc, char * argv) {
    hipDeviceProp_t deviceProp;
    handleCudaErrors(hipGetDeviceProperties(&deviceProp));

    unsigned int iterationsPerThread = 1000 * 1000;
    dim3 grid = 16;
    dim3 block = 64;

    parseArgs(argc, &argv, &iterationsPerThread, deviceProp, grid.x, block.x);
    
    
    return 0;
}
