#include <iostream>
#include <ctime>
#include <algorithm>
#include <functional>
#include <vector>
#include <numeric>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include "naive_kernel.hu"
#include "batchrng_kernel.hu"
#include "misc.hu"

double compute_naive(dim3 grid, dim3 block, unsigned int device,
		     unsigned int iterationsperThread);

double compute_batchrng(dim3 grid, dim3 block, unsigned int device,
			unsigned int iterationsperThread,
			hipDeviceProp_t *const deviceProp);

double compute_batchrng_2stream(dim3 grid, dim3 block, unsigned int device,
			unsigned int iterationsperThread,
			hipDeviceProp_t *const deviceProp);


int main (int argc, char ** argv)
{
    unsigned int kernel = 0;
    double piest;
    hipDeviceProp_t deviceProp;
    unsigned int device = 0;
    handleCudaErrors(hipGetDeviceProperties(&deviceProp, device));

    unsigned int iterationsPerThread = 1000 * 1000;
    dim3 grid = 16;
    dim3 block = 64;

    parseArgs(argc, argv, &iterationsPerThread, &deviceProp,
	      &grid.x, &block.x, &kernel);

    switch (kernel) {
    case 0:
	piest = compute_naive(grid, block, device, iterationsPerThread);
	break;
    case 1:
	piest = compute_batchrng(grid, block, device, iterationsPerThread,
				 &deviceProp);
	break;
    case 2:
	piest = compute_batchrng(grid, block, device,
					 iterationsPerThread, &deviceProp);
    }

    reportResults(piest, iterationsPerThread, grid.x, block.x, &deviceProp);

    return 0;
}

double compute_naive(dim3 grid, dim3 block, unsigned int device,
		     unsigned int iterationsperThread)
{
    handleCudaErrors(hipSetDevice(device));

    hiprandState *d_rngStates = 0;
    handleCudaErrors(hipMalloc((void **) &d_rngStates, grid.x * block.x * sizeof(hiprandState)));

    float *d_res = 0;
    handleCudaErrors(hipMalloc((void **) &d_res, grid.x * sizeof(float)));

    initRNG<<<grid, block>>>(d_rngStates, time(NULL));

    naive_kernel<<<grid, block,  block.x * sizeof(unsigned int)>>>(d_res, d_rngStates, iterationsperThread);

    std::vector<float> res(grid.x);
    handleCudaErrors(hipMemcpy(&res[0], d_res, grid.x * sizeof(float),
				hipMemcpyDeviceToHost));

    double estimate = std::accumulate(res.begin(), res.end(), 0.0);
    hipFree(d_rngStates);
    hipFree(d_res);

    return estimate;
}

double compute_batchrng(dim3 grid, dim3 block, unsigned int device,
			unsigned int its,
			hipDeviceProp_t *const deviceProp)
{
    handleCudaErrors(hipSetDevice(device));
    //Set up the RNG
    using namespace std::placeholders;
    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator, time(NULL));

    auto unifGen = std::bind(hiprandGenerateUniform, generator, _1, _2);

    //For partial results
    float *d_res = 0;
    handleCudaErrors(hipMalloc((void **) &d_res, grid.x * sizeof(float)));

    //To calculate the final result
    double runningEstimate = 0;
    
    //Random number vector allocation strategy
    unsigned int numThreads = grid.x * block.x;
    unsigned long int totalSize = sizeof(float) * its * numThreads;
    unsigned int vecSize = 128 * 1024 * 1024;
    unsigned long int remainSize = totalSize;

    float * d_angleVec = 0;
    handleCudaErrors(hipMalloc((void**) &d_angleVec, vecSize));

    float * d_distVec = 0;
    handleCudaErrors(hipMalloc((void**) &d_distVec, vecSize));


    unsigned int vecCount = vecSize / sizeof(float);
    unsigned int numRuns = 0;
    std::vector<float> res(grid.x);

    unsigned int count = 0;
    //Here we go!
    while (remainSize > sizeof(float)) {
	numRuns++;
	if (remainSize < vecSize) {
	    vecCount = remainSize / sizeof(float);
	}
	count += vecCount;
	unifGen(d_angleVec, vecCount);
	unifGen(d_distVec, vecCount);

	batchrng_kernel<<<grid, block,  block.x * sizeof(unsigned int)>>>
	    ( d_res, d_angleVec, d_distVec, vecCount);

	handleCudaErrors(hipMemcpy(&res[0], d_res, grid.x * sizeof(float),
				    hipMemcpyDeviceToHost));
	runningEstimate += std::accumulate(res.begin(), res.end(), 0.0);
	
	if (remainSize > vecSize) {
	    remainSize -= vecSize;
	}
	else {
	    break;
	}
    }
    
    hipFree(d_angleVec);
    hipFree(d_distVec);
    hipFree(d_res);
    
    return runningEstimate / numRuns;
}

double compute_batchrng_2stream(dim3 grid, dim3 block, unsigned int device,
			unsigned int its,
			hipDeviceProp_t *const deviceProp)
{
    handleCudaErrors(hipSetDevice(device));
    //Set up the RNG
    using namespace std::placeholders;
    hiprandGenerator_t generator;
    hiprandCreateGenerator(&generator, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(generator, time(NULL));

    auto unifGen = std::bind(hiprandGenerateUniform, generator, _1, _2);

    //Set up our streams
    std::vector<hipStream_t> streams (2);
    std::for_each(streams.begin(), streams.end(),
		  [] (hipStream_t stream)
		  {
		      handleCudaErrors(hipStreamCreate(&stream));
		  });
    //handleCudaErrors(hipStreamCreate(&streams[0]));
    //handleCudaErrors(hipStreamCreate(&streams[1]));
    

    //For partial results
    float *d_res = 0;
    handleCudaErrors(hipMalloc((void **) &d_res, grid.x * sizeof(float)));

    float *d_res2 = 0;
    handleCudaErrors(hipMalloc((void **) &d_res2, grid.x * sizeof(float)));

    //To calculate the final result
    double runningEstimate = 0;
    
    //Random number vector allocation strategy
    unsigned int numThreads = grid.x * block.x;
    unsigned long int totalSize = sizeof(float) * its * numThreads;
    unsigned int vecSize = 128 * 1024 * 1024;
    unsigned long int remainSize = totalSize;

    float * d_angleVec = 0;
    handleCudaErrors(hipMalloc((void**) &d_angleVec, vecSize));

    float * d_distVec = 0;
    handleCudaErrors(hipMalloc((void**) &d_distVec, vecSize));

    float * d_angleVec2 = 0;
    handleCudaErrors(hipMalloc((void**) &d_angleVec2, vecSize));

    float * d_distVec2 = 0;
    handleCudaErrors(hipMalloc((void**) &d_distVec2, vecSize));

    unsigned int vecCount = vecSize / sizeof(float);
    unsigned int numRuns = 0;
    std::vector<float> res(grid.x);

    //Here we go!
    while (remainSize > sizeof(float)) {
	numRuns++;
	if (remainSize < vecSize) {
	    vecCount = remainSize / sizeof(float);
	}

	hiprandSetStream(generator, streams[0]);
	unifGen(d_angleVec, vecCount);
	unifGen(d_distVec, vecCount);

	batchrng_kernel
	    <<<grid, block, block.x * sizeof(unsigned int), streams[0]>>>
	    ( d_res, d_angleVec, d_distVec, vecCount);

	handleCudaErrors(hipMemcpyAsync(&res[0], d_res,
					 grid.x * sizeof(float),
					 hipMemcpyDeviceToHost,
					 streams[0]));

	hiprandSetStream(generator, streams[1]);
	unifGen(d_angleVec2, vecCount);
	unifGen(d_distVec2, vecCount);

	batchrng_kernel<<<grid, block,
	    block.x * sizeof(unsigned int), streams[1]>>>
	    ( d_res2, d_angleVec2, d_distVec2, vecCount);

	runningEstimate += std::accumulate(res.begin(), res.end(), 0.0);

	hipStreamSynchronize(streams[1]);
	handleCudaErrors(hipMemcpy(&res[0], d_res2,
					 grid.x * sizeof(float),
					 hipMemcpyDeviceToHost));
	runningEstimate += std::accumulate(res.begin(), res.end(), 0.0);
	
	if (remainSize > vecSize) {
	    remainSize -= vecSize;
	}
	else {
	    break;
	}
    }
    
    hipFree(d_angleVec);
    hipFree(d_distVec);
    hipFree(d_res);
    hipFree(d_angleVec2);
    hipFree(d_distVec2);
    hipFree(d_res2);
    
    return runningEstimate / numRuns;
}
