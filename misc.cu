#include <iostream>
#include <iomanip>
#include <string>
#include <stdexcept>
#include <unistd.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>

using namespace std;

void handleCudaErrors (hipError_t cudaResult, string msg)
{
    if (cudaResult != hipSuccess) {
	msg += hipGetErrorString(cudaResult);
	throw runtime_error(msg);
    }
}

void printHelpmsg ()
{
    string helpMsg = "Usage: buffoncuda [-n <NUMINT>] [-b <BLOCKNUM>] [-t <TNUM>] [-k <KERNID>] [-d <DEVID>]\n\n";
    helpMsg += "Please remember me to finish writing this if you feel frustrated by the lack of proper documentation.\n";
    cout << helpMsg;
    exit(0);
}

void parseArgs (int argc, char ** argv, unsigned int *  iterationsPerThread,
		hipDeviceProp_t * const deviceProp, unsigned int * numBlocks,
		unsigned int *  threadsPerBlock, unsigned int * kernel, int * device)
{
    char cmdFlag;
    int candidate = 0;
    bool dFlag = 0;
    hipError_t result = hipSuccess;
    
    while((cmdFlag = getopt(argc, argv, "n:b:t:k:d:h")) != -1) {
	switch (cmdFlag)
	    {
	    case 'n':
		*iterationsPerThread = atoi(optarg);
		break;
	    case 'b':
		candidate = atoi(optarg);
		if (candidate <= 0) {
		    throw runtime_error("Number of blocks must be greater than zero");
		}
		else {
		    *numBlocks = candidate;
		}
		break;
	    case 't':
		candidate = atoi(optarg);
		if (candidate <= 0) {
		    throw runtime_error("Number of threads per block must be greater than zero.");
		}
		else if ((candidate & (candidate - 1)) != 0) {
		    throw runtime_error("Number of threads per block must be a power of two(for efficient reduction).");
		}
		else {
		    *threadsPerBlock = candidate;
		}
		break;
	    case 'k':
		candidate = atoi(optarg);
		if (candidate < 0 || candidate > 2) {
		    throw runtime_error("Kernel number must be 0, 1 or 2");
		}
		else {
		    *kernel = candidate;
		}
		break;
	    case 'd':
		candidate = atoi(optarg);
		result = hipSetDevice(candidate);
		if (result != hipSuccess) {
		    string msg("Couldn't set requested device: ");
		    msg += hipGetErrorString(result);
		    throw runtime_error(msg);
		}
		*device = candidate;
		dFlag = 1;
		break;
	    case 'h':
		printHelpmsg();
		break;
	    }
    }

    if(!dFlag){
	hipSetDevice(*device);
    }
    
    hipGetDeviceProperties(deviceProp, *device);
    
    if(*threadsPerBlock > deviceProp->maxThreadsDim[0]){
	throw runtime_error("Threads per block exceeds device maximum.");
    }
    if(*numBlocks > deviceProp->maxGridSize[0]){
	throw runtime_error("Grid size exceeds device maximum.");
    }
}

void reportResults (double estimate, unsigned int itpT,  unsigned int gridS,
		    unsigned int blockS, hipDeviceProp_t *const deviceProp, float elapsedTime)
{
    double abserr = abs(estimate - HIP_PI);
    double relerr = abserr / HIP_PI;

    cout << "      RESULTS:          " << endl;
    cout << "========================" << endl;
    cout << "Device Name:            " << deviceProp->name << endl;
    cout << "Grid Size:              " << gridS << endl;
    cout << "Block Size:             " << blockS << endl;
    cout << "Number of threads:      " << blockS * gridS << endl;
    cout << "Iterations per thread:  " << itpT << endl;
    cout << "Total iterations:       " << static_cast<double>(itpT) * blockS * gridS << endl;
    cout << "Kernel execution time:  " << elapsedTime << "s" << endl;
    cout << "PI estimate:            " << estimate << endl;
    cout << "Abolute error:          " << abserr << endl;
    cout << "Relative error:         " << relerr << endl;
    
}
