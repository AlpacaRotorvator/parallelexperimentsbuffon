#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_math_constants.h>
#include "naive_kernel.hu"

__global__ void initRNG(hiprandState * const rngStates, const unsigned int seed)
{
    // Determine thread ID
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialise the RNG
    hiprand_init(seed, tid, 0, &rngStates[tid]);
}

__device__ void draw(float &angle, float &distance, hiprandState &state)
{
    angle = cosf(hiprand_uniform(&state) * HIP_PIO2_F);
    distance = hiprand_uniform(&state) * 2;
}

__device__ unsigned int reduce_sum(unsigned int in)
{
    extern __shared__ unsigned int sdata[];

    // Perform first level of reduction:
    // - Write to shared memory
    unsigned int ltid = threadIdx.x;

    sdata[ltid] = in;
    __syncthreads();

    // Do reduction in shared mem
    for (unsigned int s = blockDim.x / 2 ; s > 0 ; s >>= 1)
    {
        if (ltid < s)
        {
            sdata[ltid] += sdata[ltid + s];
        }

        __syncthreads();
    }

    return sdata[0];
}

__global__ void naive_kernel (unsigned int *const results,
			      hiprandState *const rngStates,
			      const unsigned int numSims)
{
    // Determine thread ID
    unsigned int bid = blockIdx.x;
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    //unsigned int step = gridDim.x * blockDim.x;

    // Initialise the RNG
    hiprandState localState = rngStates[tid];

    // Count the number of draws that cross the line
    unsigned int pointsInside = 0;

    for (unsigned int i = 0; i < numSims ; i++)
    {
        float angle;
        float distance;
        draw(angle, distance, localState);

        if (distance <= angle)
        {
            pointsInside++;
        }
    }

    // Reduce within the block
    pointsInside = reduce_sum(pointsInside);

    // Store the result
    if (threadIdx.x == 0)
    {
        results[bid] = pointsInside;
    }
}
